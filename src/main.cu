#include "hip/hip_runtime.h"
#include "shaderProgram.hpp"

#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <unistd.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include "hip/hip_runtime_api.h"

// mine stuffs:
#include "Keyboard.cuh"
#include "cameraStuff.cuh"
#include "initKernels.cuh"
#include "mathHelpers.cuh"

// my le raycasting
#include "raycastKernel.cuh"

constexpr uint CU_TEX_WIDTH = 1920;
constexpr uint CU_TEX_HEIGHT = 1080;

constexpr uint SPHERE_COUNT = 1000;
constexpr uint LIGHT_COUNT = 10;
 
constexpr float MOVE_SPEED = 0.3F;
constexpr float ROT_SPEED = 0.008F;

KeyboardState keyboardState;

void glfw_error_callback(int error, const char* description);
void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods);
auto glfwSetup() -> GLFWwindow*;

auto main() -> int {
    GLFWwindow* window{};
    if (nullptr == (window = glfwSetup())) {
        return 1;
    }

    // ===
    // === SHADER STUFFS (Rect VAO, VBO, EBO)
    // ===
    float positions[] = {
        1.0F,  1.0F,  0.0F,  // top right
        1.0F,  -1.0F, 0.0F,  // bottom right
        -1.0F, -1.0F, 0.0F,  // bottom left
        -1.0F, 1.0F,  0.0F   // top left
    };

    float colors[] = {
        1.0F, 0.0F, 0.0F,  // top right
        0.0F, 1.0F, 0.0F,  // top right
        0.0F, 0.0F, 1.0F,  // top right
        1.0F, 1.0F, 1.0F   // top right
    };

    uint indices[] = {
        // note that we start from 0!
        0, 1, 3,  // first triangle
        1, 2, 3   // second triangle
    };

    // setup and bind vertex array object
    uint rectangle_VAO{};
    glGenVertexArrays(1, &rectangle_VAO);
    glBindVertexArray(rectangle_VAO);

    // setup vertex buffer object
    // with our vertices
    uint rectangle_positions_VBO{};
    glGenBuffers(1, &rectangle_positions_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_positions_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(positions),
                 static_cast<GLvoid*>(positions), GL_STATIC_DRAW);

    // define the location and format of the vertex position attribute,
    // index=0, b/c we said location=0
    // 3 b/c 3 values,
    // GL_FALSE b/c we don't need normalization,
    // 3*floatsize is stride, (0 means packed, equivalent in this case)
    // first coord is at [0]
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, nullptr);

    // setup some colors too
    uint rectangle_colors_VBO{};
    glGenBuffers(1, &rectangle_colors_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_colors_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(colors),
                 static_cast<GLvoid*>(colors), GL_STATIC_DRAW);

    // similar format for the colors
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 0, nullptr);

    // setup the element buffer object
    uint rectangle_points_EBO{};
    glGenBuffers(1, &rectangle_points_EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, rectangle_points_EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices),
                 static_cast<GLvoid*>(indices), GL_STATIC_DRAW);

    // enable the vertex attrib arrays?
    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);

    ShaderProgram leShaderProgram{
        ShaderUnit<GL_FRAGMENT_SHADER>{File{"./shaders/fragment.glsl"}},
        ShaderUnit<GL_VERTEX_SHADER>{File{"./shaders/vertex.glsl"}}
    };

    // ===
    // === TEXTURE BINDING STUFFS (BOX)
    // ===
    uint boxTexture{};
    glGenTextures(1, &boxTexture);

    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, boxTexture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    {
        int width{};
        int height{};
        int nrChannels{};

        stbi_set_flip_vertically_on_load(static_cast<int>(true));
        uint8_t* data = stbi_load("./textures/container.jpg", &width, &height, &nrChannels, 0);
        if (data != nullptr) {
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, width, height, 0, GL_RGB, GL_UNSIGNED_BYTE, data);
            glGenerateMipmap(GL_TEXTURE_2D); // probably not needed for later lol
        } else {
            std::cerr << "Failed to load texture\n";
        }

        stbi_image_free(data);
    }

    // ===
    // === TEXTURE BINDING STUFFS (SMILEY)
    // ===
    uint smileyTexture{};
    glGenTextures(1, &smileyTexture);

    glActiveTexture(GL_TEXTURE1);
    glBindTexture(GL_TEXTURE_2D, smileyTexture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    {
        int width{};
        int height{};
        int nrChannels{};

        stbi_set_flip_vertically_on_load(static_cast<int>(true));

        uint8_t* data = stbi_load("./textures/awesomeface.png", &width, &height, &nrChannels, 0);
        if (data != nullptr) {
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, data);
            glGenerateMipmap(GL_TEXTURE_2D); // probably not needed for later lol
        } else {
            std::cerr << "Failed to load texture\n";
        }

        stbi_image_free(data);
    }

    // ===
    // === TEXTURE VBO STUFFS
    // ===
    float tex_coords[] = {
        1.0F,  1.0F, // top right
        1.0F,  0.0F, // bottom right
        0.0F, 0.0F, // bottom left
        0.0F, 1.0F, // top left
    };

    uint rectangle_tex_VBO{};
    glGenBuffers(1, &rectangle_tex_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_tex_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(tex_coords),
                 static_cast<GLvoid*>(tex_coords), GL_STATIC_DRAW);

    // similar format for the colors
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 0, nullptr);
    glEnableVertexAttribArray(2);

    // ===
    // === CUDA TEXTURE STUFFS
    // ===

    // Create an OpenGL texture
    GLuint texture_id{};
    glGenTextures(1, &texture_id);
    glActiveTexture(GL_TEXTURE2);
    glBindTexture(GL_TEXTURE_2D, texture_id);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, CU_TEX_WIDTH, CU_TEX_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    // Register the texture with CUDA
    hipGraphicsResource* cuda_texture_resource{};
    checkCudaErrors(hipGraphicsGLRegisterImage(&cuda_texture_resource, texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));

    // install shader and set uniforms so we can tell samplers' offsets ig?
    leShaderProgram.glUseProgram();
    leShaderProgram.glUniform("box", 0);
    leShaderProgram.glUniform("smiley", 1);
    leShaderProgram.glUniform("cuda", 2);
    glUseProgram(0);

    // ===
    // === SPHERE / LIGHTS SETUP
    // ===
    Spheres* spheresInfo = nullptr;
    checkCudaErrors(hipMallocManaged(&spheresInfo, sizeof(Spheres)));
    checkCudaErrors(hipMalloc(&spheresInfo->centers, sizeof(float3) * SPHERE_COUNT));
    checkCudaErrors(hipMalloc(&spheresInfo->radii, sizeof(float) * SPHERE_COUNT));

    Lights* lightsInfo = nullptr;
    checkCudaErrors(hipMallocManaged(&lightsInfo, sizeof(Lights)));
    checkCudaErrors(hipMalloc(&lightsInfo->centers, sizeof(float3) * LIGHT_COUNT));
    checkCudaErrors(hipMalloc(&lightsInfo->colors, sizeof(float3) * LIGHT_COUNT));

    // Hand picked for decent (-ish) scaling up to 500 units, probably should tweak depending on how far out the lights might be
    lightsInfo->attenuation = {1, 0.001, 0.00002};
    lightsInfo->ambientStrength = 0.1 / LIGHT_COUNT;

    float3 centerMin = {5, 5, 5};
    float3 centerMax = {500, 500, 500};
    float radiusMin = 1;
    float radiusMax = 5;

    // Initialize cuRAND states
    hiprandState_t* sphereRandStates = nullptr;
    hiprandState_t* lightRandStates = nullptr;
    checkCudaErrors(hipMalloc(&sphereRandStates, SPHERE_COUNT * sizeof(hiprandState_t)));
    checkCudaErrors(hipMalloc(&lightRandStates, LIGHT_COUNT * sizeof(hiprandState_t)));

#ifdef RAND_SEED
    uint sphereSeed = RAND_SEED;
    uint lightSeed = RAND_SEED;
#else // !RAND_SEED
    uint sphereSeed = time(nullptr);
    uint lightSeed = time(nullptr) + 1;
#endif // RAND_SEED

    initRand<<<(SPHERE_COUNT + 255) / 256, 256>>>(sphereSeed, sphereRandStates); // preset seeds for easier debugging, for now
    initRand<<<(LIGHT_COUNT + 255) / 256, 256>>>(lightSeed, lightRandStates);
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize random spheres and lights
    initSpheres<<<(SPHERE_COUNT + 255) / 256, 256>>>(spheresInfo, centerMin, centerMax, radiusMin, radiusMax, SPHERE_COUNT, sphereRandStates);
    initLights<<<(LIGHT_COUNT + 255) / 256, 256>>>(lightsInfo, centerMin, centerMax, LIGHT_COUNT, lightRandStates);
    checkCudaErrors(hipDeviceSynchronize());

    // ===
    // === Cam Setup
    // ===
    CameraInfo* camInfo{};
    checkCudaErrors(hipMallocManaged(&camInfo, sizeof(CameraInfo)));
    memset(camInfo, 0, sizeof(CameraInfo));

    // TODO: get this from current reso or summink?
    camInfo->imageResolution = make_uint2(CU_TEX_WIDTH, CU_TEX_HEIGHT);
    camInfo->fovDegrees = 90;

    // ===
    // === RENDER LOOP
    // ===
    while (glfwWindowShouldClose(window) == 0) {
#ifdef DEBUG_TIMINGS
        hipEvent_t stepStartEvent = nullptr;
        hipEvent_t stepStopEvent = nullptr;
        float stepTime = 0;

        checkCudaErrors(hipEventCreate(&stepStartEvent));
        checkCudaErrors(hipEventCreate(&stepStopEvent));
        checkCudaErrors(hipEventRecord(stepStartEvent));
#endif // DEBUG_TIMINGS

        auto glfwTime = static_cast<float>(glfwGetTime());

        glClearColor(0.2F, 0.3F, 0.3F, 1.0F);
        glClear(GL_COLOR_BUFFER_BIT);

        // ===
        // ===
        // ===

        // Map the cuda texture to CUDA
        hipArray* cuda_texture_array{};
        checkCudaErrors(hipGraphicsMapResources(1, &cuda_texture_resource));
        checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&cuda_texture_array, cuda_texture_resource, 0, 0));

         // Create a surface object
        hipResourceDesc res_desc{};
        res_desc.resType = hipResourceTypeArray;
        res_desc.res.array.array = cuda_texture_array;
        hipSurfaceObject_t output_surface{};
        checkCudaErrors(hipCreateSurfaceObject(&output_surface, &res_desc));

        // === Run the CUDA kernel
        dim3 block(16, 16);
        dim3 grid((CU_TEX_WIDTH + block.x - 1) / block.x, (CU_TEX_HEIGHT + block.y - 1) / block.y);

        raycast_kernel<<<grid, block>>>(output_surface, camInfo, spheresInfo, SPHERE_COUNT, lightsInfo, LIGHT_COUNT);
        checkCudaErrors(hipDeviceSynchronize());

        // Unmap the texture so that OpenGL can use it
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_texture_resource));

        // ===
        // ===
        // ===

        // install the shader program and draw stuffs
        leShaderProgram.glUseProgram();

        // pass time to the shaders so we can have a fiesta
        leShaderProgram.glUniform("glfwTime", glfwTime);

        glBindVertexArray(rectangle_VAO);
        // glDrawArrays(GL_TRIANGLES, 0, 3); // draw 3 verts
        glDrawElements(GL_TRIANGLES, sizeof(indices), GL_UNSIGNED_INT,
                       nullptr);  // draw using ebo
        glBindVertexArray(0);     // unbind, no need to unbind it every time tho

        glfwSwapBuffers(window);

#ifdef DEBUG_TIMINGS
        checkCudaErrors(hipEventRecord(stepStopEvent));
        checkCudaErrors(hipEventSynchronize(stepStopEvent));
        checkCudaErrors(hipEventElapsedTime(&stepTime, stepStartEvent,
                                             stepStopEvent));
        std::fprintf(stderr, "%6.3fms\n", stepTime);
#endif // DEBUG_TIMINGS

        glfwPollEvents();
        camInfo->center += keyboardState.getPositionDelta() * MOVE_SPEED;
        camInfo->eulerAngles += keyboardState.getEulerDelta() * ROT_SPEED;

#ifdef PAUSE_FRAMES
        getchar(); // tmp boonk for going frame by frame
#endif // PAUSE_FRAMES
    }

    // cleanup a little and exit
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_texture_resource));
    glDeleteVertexArrays(1, &rectangle_VAO);
    glDeleteBuffers(1, &rectangle_positions_VBO);
    glDeleteBuffers(1, &rectangle_points_EBO);
    glfwDestroyWindow(window);
    glfwTerminate();
}

auto glfwSetup() -> GLFWwindow* {
    // set the error callback function for glfw stuff
    glfwSetErrorCallback(glfw_error_callback);

    // init glfw
    glfwInit();

    // hint that we'll use OpenGL 3.3 core? not sure exactly
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
#ifdef __APPLE__
    // not sure if this is even enough
    // with all the weird stuff I've done anyway lole
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

    // create a window and context, check for errors,
    // make this window current
    GLFWwindow* window = glfwCreateWindow(
        CU_TEX_WIDTH, CU_TEX_HEIGHT, "IM GLing LESSGOOO", nullptr, nullptr);
    if (window == nullptr) {
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return nullptr;
    }
    glfwMakeContextCurrent(window);

    // init the glad loader or something, not sure
    if (gladLoadGLLoader(reinterpret_cast<GLADloadproc>(glfwGetProcAddress)) == 0) {
        std::cerr << "Failed to initialize GLAD\n";
        return nullptr;
    }

    // set the viewport dimensions?
    glViewport(0, 0, CU_TEX_WIDTH, CU_TEX_HEIGHT);

    // set callbacks for window resizes and keystrokes
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwSetKeyCallback(window, key_callback);

    return window;
}

void glfw_error_callback(int error, const char* description) {
    fprintf(stderr, "Oops![0x%08X]: %s\n", error, description);
    glfwTerminate();
    exit(EXIT_FAILURE);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height) {
    (void)window;
    std::cerr << "Resized to: " << width << 'x' << height << '\n';
    glViewport(0, 0, width, height);
}

void key_callback(GLFWwindow* window, int key, int scancode, int action,
                 int mods) {
    (void)scancode;
    (void)mods;

#ifdef DEBUG_KEYBOARD
    printf("Key:%d Action:%d\n", key, action);
#endif // DEBUG_KEYBOARD

    // close if Esc pressed
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
        glfwSetWindowShouldClose(window, GLFW_TRUE);
    } else {
        keyboardState.handleKeyPress(key, action != GLFW_RELEASE);

#ifdef DEBUG_KEYBOARD
        float3 positionDelta = keyboardState.getPositionDelta(); 
        float3 eulerDelta = keyboardState.getEulerDelta(); 
        printf("Pos Delta: {%f, %f, %f}, Euler Delta: {%f, %f, %f}\n",
               positionDelta.x, positionDelta.y, positionDelta.z,
               eulerDelta.x, eulerDelta.y, eulerDelta.z
               );
#endif // DEBUG_KEYBOARD
    }
}
