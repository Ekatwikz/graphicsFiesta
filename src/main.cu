#include "hip/hip_runtime.h"
#include <cfloat>
#include "shaderProgram.hpp"

#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <unistd.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include "hip/hip_runtime_api.h"

#include "Keyboard.hpp"

constexpr uint CU_TEX_WIDTH = 1920;
constexpr uint CU_TEX_HEIGHT = 1080;

constexpr uint SPHERE_COUNT = 1000;
constexpr uint LIGHT_COUNT = 10;

constexpr float MOVE_SPEED = 0.3F;
constexpr float ROT_SPEED = 0.008F;

// callbacks
void glfw_error_callback(int error, const char* description) {
    fprintf(stderr, "Oops![0x%08X]: %s\n", error, description);
    glfwTerminate();
    exit(EXIT_FAILURE);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height) {
    (void)window;
    std::cerr << "Resized to: " << width << 'x' << height << '\n';
    glViewport(0, 0, width, height);
}

KeyboardState keyboardState;

void key_callback(GLFWwindow* window, int key, int scancode, int action,
                 int mods) {
    (void)scancode;
    (void)mods;

#ifdef DEBUG_KEYBOARD
    printf("Key:%d Action:%d\n", key, action);
#endif // DEBUG_KEYBOARD

    // close if Esc pressed
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
        glfwSetWindowShouldClose(window, GLFW_TRUE);
    } else {
        keyboardState.handleKeyPress(key, action != GLFW_RELEASE);

#ifdef DEBUG_KEYBOARD
        float3 positionDelta = keyboardState.getPositionDelta(); 
        float3 eulerDelta = keyboardState.getEulerDelta(); 
        printf("Pos Delta: {%f, %f, %f}, Euler Delta: {%f, %f, %f}\n",
               positionDelta.x, positionDelta.y, positionDelta.z,
               eulerDelta.x, eulerDelta.y, eulerDelta.z
               );
#endif // DEBUG_KEYBOARD
    }
}

__device__ __host__ auto deg2rad(float degs) -> float {
    return degs * M_PI / 180;
}

using CameraInfo = struct CameraInfo_ {
    float3 center;
    uint2 imageResolution;
    float fovDegrees; // radians?
    float3 eulerAngles;
};

// SOA type stuff ig?
struct Spheres {
    float3* centers;
    float* radii;
};
struct Lights {
    float3* centers;
    float3* colors;
    float3 attenuation;
    float ambientStrength;
};

__global__ void initRand(unsigned int seed, hiprandState_t* states) {
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void initSpheres(Spheres* spheres, float3 centerMin, float3 centerMax, float radiusMin, float radiusMax, uint numSpheres, hiprandState_t* states) {
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numSpheres) {
        hiprandState_t state = states[idx];
        spheres->centers[idx] = {
            (centerMin.x + hiprand_uniform(&state) * (centerMax.x - centerMin.x)) * (hiprand_uniform(&state) > 0.5F ? 1 : -1),
            (centerMin.y + hiprand_uniform(&state) * (centerMax.y - centerMin.y)) * (hiprand_uniform(&state) > 0.5F ? 1 : -1),
            (centerMin.z + hiprand_uniform(&state) * (centerMax.z - centerMin.z)) * (hiprand_uniform(&state) > 0.5F ? 1 : -1)
        };

        spheres->radii[idx] = radiusMin + hiprand_uniform(&state) * (radiusMax - radiusMin);

#ifdef PAUSE_FRAMES
        printf("[%d]: C_S:{%lf,%lf,%lf}|R:%lf\n", idx,
               spheres->centers[idx].x, spheres->centers[idx].y, spheres->centers[idx].z,
               spheres->radii[idx]);
#endif // PAUSE_FRAMES
    }
}

__global__ void initLights(Lights* lights, float3 centerMin, float3 centerMax,  uint numLights, hiprandState_t* states) {
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numLights) {
        hiprandState_t state = states[idx];
        lights->centers[idx] = {
            (centerMin.x + hiprand_uniform(&state) * (centerMax.x - centerMin.x)) * (hiprand_uniform(&state) > 0.5F ? 1 : -1),
            (centerMin.y + hiprand_uniform(&state) * (centerMax.y - centerMin.y)) * (hiprand_uniform(&state) > 0.5F ? 1 : -1),
            (centerMin.z + hiprand_uniform(&state) * (centerMax.z - centerMin.z)) * (hiprand_uniform(&state) > 0.5F ? 1 : -1)
        };

        lights->colors[idx] = {
            hiprand_uniform(&state),
            hiprand_uniform(&state),
            hiprand_uniform(&state)
        };

#ifdef PAUSE_FRAMES
        printf("[%d]: C_L:{%lf,%lf,%lf}, C:{%f, %f, %f}\n", idx,
               lights->centers[idx].x, lights->centers[idx].y, lights->centers[idx].z,
               lights->colors[idx].x, lights->colors[idx].y, lights->colors[idx].z) ;
#endif // PAUSE_FRAMES
    }
}

struct Matrix4x4f {
    float4 rows[4];

    __device__ __host__ Matrix4x4f(const float4& row1, const float4& row2, const float4& row3, const float4& row4) {
        rows[0] = row1;
        rows[1] = row2;
        rows[2] = row3;
        rows[3] = row4;
    }

    __device__ __host__ Matrix4x4f() : Matrix4x4f{{1, 0, 0, 0}, {0, 1, 0, 0}, {0, 0, 1, 0}, {0, 0, 0, 1}} {}

    __device__ __host__ Matrix4x4f(const float3& euler_angles, const float3& camera_position) : Matrix4x4f{} {
        //  https://en.wikipedia.org/wiki/Euler_angles#Rotation_matrix
        float c_1 = cosf(euler_angles.x);
        float s_1 = sinf(euler_angles.x);
        float c_2 = cosf(euler_angles.y);
        float s_2 = sinf(euler_angles.y);
        float c_3 = cosf(euler_angles.z);
        float s_3 = sinf(euler_angles.z);

        rows[0] = make_float4(c_1 * c_2 * c_3 - s_1 * s_3, -c_3 * s_1 - c_1 * c_2 * s_3, c_1 * s_2, camera_position.x);
        rows[1] = make_float4(c_1 * s_3 + c_2 * c_3 * s_1, c_1 * c_3 - c_2 * s_1 * s_3, s_1 * s_2, camera_position.y);
        rows[2] = make_float4(-c_3 * s_2, s_2 * s_3, c_2, camera_position.z);
    }

    __inline__ __device__ __host__ auto operator*(const float4& vec) const -> float4 {
        float4 result;
        result.x = rows[0].x * vec.x + rows[0].y * vec.y + rows[0].z * vec.z + rows[0].w * vec.w;
        result.y = rows[1].x * vec.x + rows[1].y * vec.y + rows[1].z * vec.z + rows[1].w * vec.w;
        result.z = rows[2].x * vec.x + rows[2].y * vec.y + rows[2].z * vec.z + rows[2].w * vec.w;
        result.w = rows[3].x * vec.x + rows[3].y * vec.y + rows[3].z * vec.z + rows[3].w * vec.w;
        return result;
    }

    __inline__  __device__ __host__ auto operator*(const float3& vec) const -> float3 {
        float4 result = *this * make_float4(vec.x, vec.y, vec.z, 1);
        return make_float3(result.x, result.y, result.z);
    }

    // b/c they're referring to different printfs?
#define PRINT_MATRIX() do { \
        printf("{"); \
        for (const auto& row : rows) { \
            printf("{%f %f %f %f},\n", row.x, row.y, row.z, row.w); \
        } \
        printf("}"); \
} while(0)

    __device__ void d_display() const { PRINT_MATRIX(); }

    __host__ void h_display() const { PRINT_MATRIX(); }
};

auto __device__ __host__ operator-(const float3& lhs, const float3& rhs) -> float3 {
    return {lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z};
}

auto __device__ __host__ operator+(const float3& lhs, const float3& rhs) -> float3 {
    return {lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z};
}

auto __device__ __host__ operator+=(float3& lhs, const float3& rhs) -> float3 {
    return lhs = lhs + rhs;
}

auto __device__ __host__ dot(const float3& lhs, const float3& rhs) -> float {
    return lhs.x * rhs.x + lhs.y * rhs.y + lhs.z * rhs.z;
}

// pure spaghetti code, yikes
// TODO: template-ize this crap
auto __device__ __host__ clamp(float3& vec, float min, float max) -> float3 {
    if (vec.x < min) {
        vec.x = min;
    } else if (vec.x > max) {
        vec.x = max;
    }

    if (vec.y < min) {
        vec.y = min;
    } else if (vec.y > max) {
        vec.y = max;
    }

    if (vec.z < min) {
        vec.z = min;
    } else if (vec.z > max) {
        vec.z = max;
    }

    return vec;
}

auto __device__ __host__ operator/(const float3& lhs, float rhs) -> float3 {
    return {lhs.x / rhs, lhs.y / rhs, lhs.z / rhs};
}

auto __device__ __host__ operator*(const float3& lhs, float rhs) -> float3 {
    return {lhs.x * rhs, lhs.y * rhs, lhs.z * rhs};
}

auto __device__ __host__ operator-(const float3& vec) -> float3 {
    return vec * -1.0F;
}

auto __device__ __host__ abs(const float3& vec) -> float {
    return sqrt(dot(vec, vec));
}

auto __device__ __host__ normalize(const float3& vec) -> float3 {
    return vec / abs(vec);
}

// yoinked from here: https://registry.khronos.org/OpenGL-Refpages/gl4/html/reflect.xhtml
// hopefully it just does what it says on the box (lhs reflected through rhs ig?) and I didn't mess up (💀💀)
auto __device__ __host__ reflect(const float3& lhs, const float3& rhs) -> float3 {
    return lhs - rhs * dot(lhs, rhs) * 2.0F;
}

__global__ void write_texture_kernel(hipSurfaceObject_t output_surface, CameraInfo* camInfo,
                                     Spheres* spheresInfo, uint sphereCount,
                                     Lights* lightsInfo, uint lightsCount) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint width = camInfo->imageResolution.x;
    uint height = camInfo->imageResolution.y;

    float fovScale = tan(deg2rad(camInfo->fovDegrees / 2));

    if (x < width && y < height) {
        uint2 pixel = make_uint2(x, y);
        double2 pixelNDC = make_double2((pixel.x + 0.5) / width, (pixel.y + 0.5) / height);

        float aspectRatio = 1.0F * width / height;
        float2 pixCam = make_float2((2 * pixelNDC.x - 1) * aspectRatio * fovScale,
                                      (1 - 2 * pixelNDC.y) * fovScale);
        float3 pixCamCoord = make_float3(pixCam.x, pixCam.y, -1);

        Matrix4x4f camToWorld{camInfo->eulerAngles, camInfo->center};
        float3 pixWorldCoord = camToWorld * pixCamCoord;
        float3 rayDirection = pixWorldCoord - camInfo->center;
        float3 D = normalize(rayDirection);

#ifdef PAUSE_FRAMES
        //camToWorld.d_display();
        printf("Pix:{%d,%d}|NDC:{%lf,%lf}|Cam:{%lf,%lf}|World:{%lf,%lf,%lf}|Dir:{%lf,%lf,%lf}\n",
               pixel.x, pixel.y,
               pixelNDC.x, pixelNDC.y,
               pixCam.x, pixCam.y,
               pixWorldCoord.x, pixWorldCoord.y, pixWorldCoord.z,
               rayDirection.x, rayDirection.y, rayDirection.z);
#endif // PAUSE_FRAMES

        float t_hc = NAN;
        float t_0 = FLT_MAX;
        long intersectedIndex = -1; // will just use -1 for "no intersection"

        for (uint i = 0; i < sphereCount; ++i) {
            // Geometric solution from:
            // https://github.com/scratchapixel/website/blob/main/lessons/3d-basic-rendering/minimal-ray-tracer-rendering-simple-shapes/ray-sphere-intersection.md?plain=1
            float currentRadius = spheresInfo->radii[i];

            float3 L = spheresInfo->centers[i] - camInfo->center;
            float t_ca = dot(L, D);
            if (t_ca < 0) { // sphere is behind
                continue;
            }

            float d = sqrt(dot(L, L) - t_ca * t_ca);
            if (d > spheresInfo->radii[i]) { // ray "missed"
                continue;
            }

            float curr_t_hc = sqrt(currentRadius * currentRadius - d * d);
            float curr_t_0 = t_ca - curr_t_hc;
            if (curr_t_0 > t_0) { // blocked by some closer sphere
                continue;
            }

            t_hc = curr_t_hc;
            intersectedIndex = i;
            t_0 = curr_t_0;
        }

        // === DRAW STUFFS ===
        uchar4 tmpPixData = make_uchar4(0, 0, 0, 255);

        if (-1 != intersectedIndex) { // if we're lookin at somethin
            // Lighting Setup from here:
            // https://learnopengl.com/Lighting/Basic-Lighting
            float3 intersectionPoint = D * t_0 + camInfo->center;
            float3 intersectionNormal = normalize(intersectionPoint - spheresInfo->centers[intersectedIndex]);

#ifdef PAUSE_FRAMES
            printf("%d,%d: t_hc:%lf t_0:%f r:%f [%ld] {%f, %f, %f}->{%f, %f, %f} (C_S:{%f, %f, %f})\n", x, y, t_hc, t_0,
                   spheresInfo->radii[intersectedIndex], intersectedIndex,
                   pixWorldCoord.x, pixWorldCoord.y, pixWorldCoord.z,
                   intersectionPoint.x, intersectionPoint.y, intersectionPoint.z,
                   spheresInfo->centers[intersectedIndex].x, spheresInfo->centers[intersectedIndex].y, spheresInfo->centers[intersectedIndex].z);
#endif // PAUSE_FRAMES

            float3 ambient = {0, 0, 0};
            float3 diffuse = {0, 0, 0};
            float3 specular = {0, 0, 0};
            for(uint i = 0; i < lightsCount; ++i) {
                float3 lightPos = lightsInfo->centers[i];
                float3 lightColor = lightsInfo->colors[i];
                float3 lightVec = lightPos - intersectionPoint;
                float3 lightDir = normalize(lightVec);
                float lightDistance = abs(lightVec);

                ambient += lightColor * lightsInfo->ambientStrength;

                float attenuation = 1 / (lightsInfo->attenuation.x
                    + lightsInfo->attenuation.y * lightDistance
                    + lightsInfo->attenuation.z * lightDistance * lightDistance);

                float diffuseStrength = max(dot(intersectionNormal, lightDir), 0.0F);
                diffuse += lightColor * diffuseStrength * attenuation;

                float specularIntensity = 0.5;
                float3 viewDir = normalize(camInfo->center - intersectionPoint);
                float3 reflectDir = reflect(-lightDir, intersectionNormal);
                float shininess = 32; // TODO: move this somewhere else?
                auto specularStrength = static_cast<float>(pow( max(dot(viewDir, reflectDir), 0.0F), shininess));
                specular += lightColor * specularIntensity * specularStrength * attenuation;
            }

            float3 color = (ambient + diffuse + specular) * 255;
            color = clamp(color, 0, 255); // just in case lol, idk

#ifdef PAUSE_FRAMES
            printf("A:{%f, %f, %f}|D:{%f, %f, %f}|S:{%f, %f, %f}|C:{%f, %f, %f}\n",
                   ambient.x, ambient.y, ambient.z,
                   diffuse.x, diffuse.y, diffuse.z,
                   specular.x, specular.y, specular.z,
                   color.x, color.y, color.z);
#endif // PAUSE_FRAMES

            tmpPixData = make_uchar4(color.x, color.y, color.z, 255);
        }

        // manually flip texture at the last moment, I dont remember why but opengl goofs up the texture otherwise
        surf2Dwrite(tmpPixData, output_surface, x * sizeof(uchar4), height - 1 - y);
    }
}

#define GLCHECK() (__extension__({\
    GLenum glErrorVal; \
    const char* glErrorName; \
    while ((glErrorVal = glGetError()) != GL_NO_ERROR) { \
        switch (glErrorVal) { \
            case GL_INVALID_ENUM: \
                glErrorName = TO_STR(GL_INVALID_ENUM); \
                break; \
            case GL_INVALID_VALUE: \
                glErrorName = TO_STR(GL_INVALID_VALUE); \
                break; \
            case GL_INVALID_OPERATION: \
                glErrorName = TO_STR(GL_INVALID_OPERATION); \
                break; \
            /* case GL_STACK_OVERFLOW: \
                glErrorName = TO_STR(GL_STACK_OVERFLOW); \
                break; \
            case GL_STACK_UNDERFLOW: \
                glErrorName = TO_STR(GL_STACK_UNDERFLOW); \
                break; */ \
            case GL_OUT_OF_MEMORY: \
                glErrorName = TO_STR(GL_OUT_OF_MEMORY); \
                break; \
            case GL_INVALID_FRAMEBUFFER_OPERATION: \
                glErrorName = TO_STR(GL_INVALID_FRAMEBUFFER_OPERATION); \
                break; \
            /* case GL_CONTEXT_LOST: \
                 glErrorName = TO_STR(GL_CONTEXT_LOST); \
                 break; \
             case GL_TABLE_TOO_LARGE: \
                 break; \
                 glErrorName = TO_STR(GL_TABLE_TOO_LARGE); */ \
            default: \
                glErrorName = "???"; \
                break; \
        } \
        fprintf(stderr, __FILE__ ":%d in %s | glGetError()->0x%08X (%s)\n", __LINE__, static_cast<const char*>(__func__), glErrorVal, glErrorName); \
    } \
    glErrorVal; \
}))

auto main() -> int {
    // ===
    // === GLFW STUFFS
    // ===

    // set the error callback function for glfw stuff
    glfwSetErrorCallback(glfw_error_callback);

    // init glfw
    glfwInit();

    // hint that we'll use OpenGL 3.3 core? not sure exactly
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
#ifdef __APPLE__
    // lole
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

    // create a window and context, check for errors,
    // make this window current
    GLFWwindow* window = glfwCreateWindow(
        CU_TEX_WIDTH, CU_TEX_HEIGHT, "IM GLing LESSGOOO", nullptr, nullptr);
    if (window == nullptr) {
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return 1;
    }
    glfwMakeContextCurrent(window);

    // init the glad loader or something, not sure
    if (gladLoadGLLoader(reinterpret_cast<GLADloadproc>(glfwGetProcAddress)) ==
        0) {
        std::cerr << "Failed to initialize GLAD\n";
        return 1;
    }

    // set the viewport dimensions?
    glViewport(0, 0, CU_TEX_WIDTH, CU_TEX_HEIGHT);

    // set callbacks for window resizes and keystrokes
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwSetKeyCallback(window, key_callback);

    // ===
    // === SHADER STUFFS (Rect VAO, VBO, EBO)
    // ===
    float positions[] = {
        1.0F,  1.0F,  0.0F,  // top right
        1.0F,  -1.0F, 0.0F,  // bottom right
        -1.0F, -1.0F, 0.0F,  // bottom left
        -1.0F, 1.0F,  0.0F   // top left
    };

    float colors[] = {
        1.0F, 0.0F, 0.0F,  // top right
        0.0F, 1.0F, 0.0F,  // top right
        0.0F, 0.0F, 1.0F,  // top right
        1.0F, 1.0F, 1.0F   // top right
    };

    uint indices[] = {
        // note that we start from 0!
        0, 1, 3,  // first triangle
        1, 2, 3   // second triangle
    };

    // setup and bind vertex array object
    uint rectangle_VAO;
    glGenVertexArrays(1, &rectangle_VAO);
    glBindVertexArray(rectangle_VAO);

    // setup vertex buffer object
    // with our vertices
    uint rectangle_positions_VBO;
    glGenBuffers(1, &rectangle_positions_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_positions_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(positions),
                 static_cast<GLvoid*>(positions), GL_STATIC_DRAW);

    // define the location and format of the vertex position attribute,
    // index=0, b/c we said location=0
    // 3 b/c 3 values,
    // GL_FALSE b/c we don't need normalization,
    // 3*floatsize is stride, (0 means packed, equivalent in this case)
    // first coord is at [0]
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, nullptr);

    // setup some colors too
    uint rectangle_colors_VBO;
    glGenBuffers(1, &rectangle_colors_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_colors_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(colors),
                 static_cast<GLvoid*>(colors), GL_STATIC_DRAW);

    // similar format for the colors
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 0, nullptr);

    // setup the element buffer object
    uint rectangle_points_EBO = 0;
    glGenBuffers(1, &rectangle_points_EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, rectangle_points_EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices),
                 static_cast<GLvoid*>(indices), GL_STATIC_DRAW);

    // enable the vertex attrib arrays?
    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);

    ShaderProgram leShaderProgram{
        ShaderUnit<GL_FRAGMENT_SHADER>{File{"./shaders/fragment.glsl"}},
        ShaderUnit<GL_VERTEX_SHADER>{File{"./shaders/vertex.glsl"}}
    };

    // ===
    // === TEXTURE BINDING STUFFS (BOX)
    // ===
    unsigned int boxTexture;
    glGenTextures(1, &boxTexture);

    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, boxTexture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    {
        int width, height, nrChannels;
        stbi_set_flip_vertically_on_load(static_cast<int>(true));
        uint8_t* data = stbi_load("./textures/container.jpg", &width, &height, &nrChannels, 0);
        if (data != nullptr) {
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, width, height, 0, GL_RGB, GL_UNSIGNED_BYTE, data);
            glGenerateMipmap(GL_TEXTURE_2D); // probably not needed for later lol
        } else {
            std::cerr << "Failed to load texture\n";
        }

        stbi_image_free(data);
    }

    // ===
    // === TEXTURE BINDING STUFFS (SMILEY)
    // ===
    unsigned int smileyTexture;
    glGenTextures(1, &smileyTexture);

    glActiveTexture(GL_TEXTURE1);
    glBindTexture(GL_TEXTURE_2D, smileyTexture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    {
        int width, height, nrChannels;
        stbi_set_flip_vertically_on_load(static_cast<int>(true));
        uint8_t* data = stbi_load("./textures/awesomeface.png", &width, &height, &nrChannels, 0);
        if (data != nullptr) {
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, data);
            glGenerateMipmap(GL_TEXTURE_2D); // probably not needed for later lol
        } else {
            std::cerr << "Failed to load texture\n";
        }

        stbi_image_free(data);
    }

    // ===
    // === TEXTURE VBO STUFFS
    // ===
    float tex_coords[] = {
        1.0F,  1.0F, // top right
        1.0F,  0.0F, // bottom right
        0.0F, 0.0F, // bottom left
        0.0F, 1.0F, // top left
    };

    uint rectangle_tex_VBO;
    glGenBuffers(1, &rectangle_tex_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_tex_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(tex_coords),
                 static_cast<GLvoid*>(tex_coords), GL_STATIC_DRAW);

    // similar format for the colors
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 0, nullptr);
    glEnableVertexAttribArray(2);

    // ===
    // === CUDA TEXTURE STUFFS
    // ===

    // Create an OpenGL texture
    GLuint texture_id;
    glGenTextures(1, &texture_id);
    glActiveTexture(GL_TEXTURE2);
    glBindTexture(GL_TEXTURE_2D, texture_id);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, CU_TEX_WIDTH, CU_TEX_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    // Register the texture with CUDA
    hipGraphicsResource* cuda_texture_resource;
    checkCudaErrors(hipGraphicsGLRegisterImage(&cuda_texture_resource, texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));

    // install shader and set uniforms so we can tell samplers' offsets ig?
    leShaderProgram.glUseProgram();
    leShaderProgram.glUniform("box", 0);
    leShaderProgram.glUniform("smiley", 1);
    leShaderProgram.glUniform("cuda", 2);
    glUseProgram(0);

    // ===
    // === SPHERE / LIGHTS SETUP
    // ===
    Spheres* spheresInfo = nullptr;
    checkCudaErrors(hipMallocManaged(&spheresInfo, sizeof(Spheres)));
    checkCudaErrors(hipMalloc(&spheresInfo->centers, sizeof(float3) * SPHERE_COUNT));
    checkCudaErrors(hipMalloc(&spheresInfo->radii, sizeof(float) * SPHERE_COUNT));
    Lights* lightsInfo = nullptr;
    checkCudaErrors(hipMallocManaged(&lightsInfo, sizeof(Lights)));
    checkCudaErrors(hipMalloc(&lightsInfo->centers, sizeof(float3) * LIGHT_COUNT));
    checkCudaErrors(hipMalloc(&lightsInfo->colors, sizeof(float3) * LIGHT_COUNT));

    // Hand picked for decent (-ish) scaling up to 500 units, probably should tweak depending on how far out the lights might be
    lightsInfo->attenuation = {1, 0.001, 0.00002};
    lightsInfo->ambientStrength = 0.1 / LIGHT_COUNT;

    float3 centerMin = {5, 5, 5};
    float3 centerMax = {500, 500, 500};
    float radiusMin = 1;
    float radiusMax = 5;

    // Initialize cuRAND states
    hiprandState_t* sphereRandStates = nullptr;
    hiprandState_t* lightRandStates = nullptr;
    checkCudaErrors(hipMalloc(&sphereRandStates, SPHERE_COUNT * sizeof(hiprandState_t)));
    checkCudaErrors(hipMalloc(&lightRandStates, LIGHT_COUNT * sizeof(hiprandState_t)));

#ifdef RAND_SEED
    uint sphereSeed = RAND_SEED;
    uint lightSeed = RAND_SEED;
#else // !RAND_SEED
    uint sphereSeed = time(nullptr);
    uint lightSeed = time(nullptr) + 1;
#endif // RAND_SEED

    initRand<<<(SPHERE_COUNT + 255) / 256, 256>>>(sphereSeed, sphereRandStates); // preset seeds for easier debugging, for now
    initRand<<<(LIGHT_COUNT + 255) / 256, 256>>>(lightSeed, lightRandStates);
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize random spheres and lights
    initSpheres<<<(SPHERE_COUNT + 255) / 256, 256>>>(spheresInfo, centerMin, centerMax, radiusMin, radiusMax, SPHERE_COUNT, sphereRandStates);
    initLights<<<(LIGHT_COUNT + 255) / 256, 256>>>(lightsInfo, centerMin, centerMax, LIGHT_COUNT, lightRandStates);
    checkCudaErrors(hipDeviceSynchronize());

    // ===
    // === Cam Setup
    // ===
    CameraInfo* camInfo = nullptr;

    checkCudaErrors(hipMallocManaged(&camInfo, sizeof(CameraInfo)));
    memset(camInfo, 0, sizeof(CameraInfo));

    // TODO: get this from current reso or summink?
    camInfo->imageResolution = make_uint2(CU_TEX_WIDTH, CU_TEX_HEIGHT);
    camInfo->fovDegrees = 90;

    // ===
    // === RENDER LOOP
    // ===
    while (glfwWindowShouldClose(window) == 0) {
#ifdef DEBUG_TIMINGS
        hipEvent_t stepStartEvent = nullptr;
        hipEvent_t stepStopEvent = nullptr;
        float stepTime = 0;

        checkCudaErrors(hipEventCreate(&stepStartEvent));
        checkCudaErrors(hipEventCreate(&stepStopEvent));
        checkCudaErrors(hipEventRecord(stepStartEvent));
#endif // DEBUG_TIMINGS

        auto glfwTime = static_cast<float>(glfwGetTime());

        glClearColor(0.2F, 0.3F, 0.3F, 1.0F);
        glClear(GL_COLOR_BUFFER_BIT);

        // ===
        // ===
        // ===

        // Map the cuda texture to CUDA
        hipArray* cuda_texture_array;
        checkCudaErrors(hipGraphicsMapResources(1, &cuda_texture_resource));
        checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&cuda_texture_array, cuda_texture_resource, 0, 0));

         // Create a surface object
        hipResourceDesc res_desc;
        memset(&res_desc, 0, sizeof(res_desc));
        res_desc.resType = hipResourceTypeArray;
        res_desc.res.array.array = cuda_texture_array;
        hipSurfaceObject_t output_surface;
        checkCudaErrors(hipCreateSurfaceObject(&output_surface, &res_desc));

        // === Run the CUDA kernel
        dim3 block(16, 16);
        dim3 grid((CU_TEX_WIDTH + block.x - 1) / block.x, (CU_TEX_HEIGHT + block.y - 1) / block.y);

        write_texture_kernel<<<grid, block>>>(output_surface, camInfo, spheresInfo, SPHERE_COUNT, lightsInfo, LIGHT_COUNT);
        checkCudaErrors(hipDeviceSynchronize());

        // Unmap the texture so that OpenGL can use it
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_texture_resource));

        // ===
        // ===
        // ===

        // install the shader program and draw stuffs
        leShaderProgram.glUseProgram();

        // pass time to the shaders so we can have a fiesta
        leShaderProgram.glUniform("glfwTime", glfwTime);

        glBindVertexArray(rectangle_VAO);
        // glDrawArrays(GL_TRIANGLES, 0, 3); // draw 3 verts
        glDrawElements(GL_TRIANGLES, sizeof(indices), GL_UNSIGNED_INT,
                       nullptr);  // draw using ebo
        glBindVertexArray(0);     // unbind, no need to unbind it every time tho

        glfwSwapBuffers(window);

#ifdef DEBUG_TIMINGS
        checkCudaErrors(hipEventRecord(stepStopEvent));
        checkCudaErrors(hipEventSynchronize(stepStopEvent));
        checkCudaErrors(hipEventElapsedTime(&stepTime, stepStartEvent,
                                             stepStopEvent));
        std::fprintf(stderr, "%6.3fms\n", stepTime);
#endif // DEBUG_TIMINGS

        glfwPollEvents();
        camInfo->center += keyboardState.getPositionDelta() * MOVE_SPEED;
        camInfo->eulerAngles += keyboardState.getEulerDelta() * ROT_SPEED;

#ifdef PAUSE_FRAMES
        getchar(); // tmp boonk for going frame by frame
#endif // PAUSE_FRAMES
    }

    // cleanup a little and exit
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_texture_resource));
    glDeleteVertexArrays(1, &rectangle_VAO);
    glDeleteBuffers(1, &rectangle_positions_VBO);
    glDeleteBuffers(1, &rectangle_points_EBO);
    glfwDestroyWindow(window);
    glfwTerminate();
}
