#include "hip/hip_runtime.h"
#include "shaderProgram.hpp"

#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <unistd.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include "hip/hip_runtime_api.h"

// callbacks
void error_callback(int error, const char* description) {
    fprintf(stderr, "Oops![0x%08X]: %s\n", error, description);
    glfwTerminate();
    exit(EXIT_FAILURE);
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height) {
    (void)window;
    std::cerr << "Resized to: " << width << 'x' << height << '\n';
    glViewport(0, 0, width, height);
}

void key_handler(GLFWwindow* window, int key, int scancode, int action,
                 int mods) {
    (void)scancode;
    (void)mods;

    printf("Key:%d Action:%d\n", key, action);

    // close if Esc pressed
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
        glfwSetWindowShouldClose(window, GLFW_TRUE);
    }
}

using CameraInfo = struct CameraInfo_ {
    float3 center;
    uint2 imageResolution;
    float fovDegrees; // radians?
    float3 eulerAngles;
};

__device__ __host__ auto deg2rad(float degs) -> float {
    return degs * M_PI / 180;
}

struct Matrix4x4f {
    float4 rows[4];

    __device__ __host__ Matrix4x4f(const float4& row1, const float4& row2, const float4& row3, const float4& row4) {
        rows[0] = row1;
        rows[1] = row2;
        rows[2] = row3;
        rows[3] = row4;
    }

    __device__ __host__ Matrix4x4f() : Matrix4x4f{{1, 0, 0, 0}, {0, 1, 0, 0}, {0, 0, 1, 0}, {0, 0, 0, 1}} {}

    __device__ __host__ Matrix4x4f(const float3& euler_angles, const float3& camera_position) : Matrix4x4f{} {
        //  https://en.wikipedia.org/wiki/Euler_angles#Rotation_matrix
        float c_1 = cosf(euler_angles.x);
        float s_1 = sinf(euler_angles.x);
        float c_2 = cosf(euler_angles.y);
        float s_2 = sinf(euler_angles.y);
        float c_3 = cosf(euler_angles.z);
        float s_3 = sinf(euler_angles.z);

        rows[0] = make_float4(c_1 * c_2 * c_3 - s_1 * s_3, -c_3 * s_1 - c_1 * c_2 * s_3, c_1 * s_2, camera_position.x);
        rows[1] = make_float4(c_1 * s_3 + c_2 * c_3 * s_1, c_1 * c_3 - c_2 * s_1 * s_3, s_1 * s_2, camera_position.y);
        rows[2] = make_float4(-c_3 * s_2, s_2 * s_3, c_2, camera_position.z);
    }

    __device__ __host__ auto operator*(const float4& vec) const -> float4 {
        float4 result;
        result.x = rows[0].x * vec.x + rows[0].y * vec.y + rows[0].z * vec.z + rows[0].w * vec.w;
        result.y = rows[1].x * vec.x + rows[1].y * vec.y + rows[1].z * vec.z + rows[1].w * vec.w;
        result.z = rows[2].x * vec.x + rows[2].y * vec.y + rows[2].z * vec.z + rows[2].w * vec.w;
        result.w = rows[3].x * vec.x + rows[3].y * vec.y + rows[3].z * vec.z + rows[3].w * vec.w;
        return result;
    }

    __device__ __host__ auto operator*(const float3& vec) const -> float3 {
        float4 result = *this * make_float4(vec.x, vec.y, vec.z, 1);
        return make_float3(result.x, result.y, result.z);
    }

    __device__ void d_display() const {
        for (const auto& row : rows) {
            printf("%f %f %f %f\n", row.x, row.y, row.z, row.w);
        }
    }

    __host__ void h_display() const {
        for (const auto& row : rows) {
            printf("%f %f %f %f\n", row.x, row.y, row.z, row.w);
        }
    }
};

auto __device__ __host__ operator-(const float3& lhs, const float3& rhs) -> float3 {
    return make_float3(lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z);
}

__global__ void write_texture_kernel(hipSurfaceObject_t output_surface, CameraInfo* camInfo, float glfwTime) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint width = camInfo->imageResolution.x;
    uint height = camInfo->imageResolution.y;

    float fovScale = tan(deg2rad(camInfo->fovDegrees / 2));

    if (x < width && y < height) {
        uint2 pixel = make_uint2(x, y);
        double2 pixelNDC = make_double2((pixel.x + 0.5) / width, (pixel.y + 0.5) / height);

        float aspectRatio = 1.0F * width / height;
        float2 pixCam = make_float2((2 * pixelNDC.x - 1) * aspectRatio * fovScale,
                                      (1 - 2 * pixelNDC.y) * fovScale);
        float3 pixCamCoord = make_float3(pixCam.x, pixCam.y, -1);

        Matrix4x4f camToWorld{camInfo->eulerAngles, camInfo->center};
        float3 pixWorldCoord = camToWorld * pixCamCoord;
        float3 rayDirection = pixWorldCoord - camInfo->center;

#ifdef PAUSE_FRAMES
        //camToWorld.d_display();
        printf("Pix:%d,%d|NDC:%lf,%lf|Cam:%lf,%lf|World:%lf,%lf,%lf|Dir:%lf,%lf,%lf\n",
               pixel.x, pixel.y,
               pixelNDC.x, pixelNDC.y,
               pixCam.x, pixCam.y,
               pixWorldCoord.x, pixWorldCoord.y, pixWorldCoord.z,
               rayDirection.x, rayDirection.y, rayDirection.z);
#endif // PAUSE_FRAMES

        uchar4 tmpPixData = make_uchar4((x + (int)(glfwTime * 3)) * 100 % 256, (y + (int)(glfwTime)) * 100 % 256, 0, 255);
        surf2Dwrite(tmpPixData, output_surface, x * sizeof(uchar4), y);
    }
}

#define GLCHECK() (__extension__({\
    GLenum glErrorVal; \
    const char* glErrorName; \
    while ((glErrorVal = glGetError()) != GL_NO_ERROR) { \
        switch (glErrorVal) { \
            case GL_INVALID_ENUM: \
                glErrorName = TO_STR(GL_INVALID_ENUM); \
                break; \
            case GL_INVALID_VALUE: \
                glErrorName = TO_STR(GL_INVALID_VALUE); \
                break; \
            case GL_INVALID_OPERATION: \
                glErrorName = TO_STR(GL_INVALID_OPERATION); \
                break; \
            /* case GL_STACK_OVERFLOW: \
                glErrorName = TO_STR(GL_STACK_OVERFLOW); \
                break; \
            case GL_STACK_UNDERFLOW: \
                glErrorName = TO_STR(GL_STACK_UNDERFLOW); \
                break; */ \
            case GL_OUT_OF_MEMORY: \
                glErrorName = TO_STR(GL_OUT_OF_MEMORY); \
                break; \
            case GL_INVALID_FRAMEBUFFER_OPERATION: \
                glErrorName = TO_STR(GL_INVALID_FRAMEBUFFER_OPERATION); \
                break; \
            /* case GL_CONTEXT_LOST: \
                 glErrorName = TO_STR(GL_CONTEXT_LOST); \
                 break; \
             case GL_TABLE_TOO_LARGE: \
                 break; \
                 glErrorName = TO_STR(GL_TABLE_TOO_LARGE); */ \
            default: \
                glErrorName = "???"; \
                break; \
        } \
        fprintf(stderr, __FILE__ ":%d in %s | glGetError()->0x%08X (%s)\n", __LINE__, __func__, glErrorVal, glErrorName); \
    } \
    glErrorVal; \
}))

constexpr uint SCR_WIDTH = 800;
constexpr uint SCR_HEIGHT = 600;

constexpr uint CU_TEX_WIDTH = 16;
constexpr uint CU_TEX_HEIGHT = 9;
auto main() -> int {
    // ===
    // === GLFW STUFFS
    // ===

    // set the error callback function for glfw stuff
    glfwSetErrorCallback(error_callback);

    // init glfw
    glfwInit();

    // hint that we'll use OpenGL 3.3 core? not sure exactly
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
#ifdef __APPLE__
    // lole
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

    // create a window and context, check for errors,
    // make this window current
    GLFWwindow* window = glfwCreateWindow(
        SCR_WIDTH, SCR_HEIGHT, "IM GLing LESSGOOO", nullptr, nullptr);
    if (window == nullptr) {
        std::cerr << "Failed to create GLFW window\n";
        glfwTerminate();
        return 1;
    }
    glfwMakeContextCurrent(window);

    // init the glad loader or something, not sure
    if (gladLoadGLLoader(reinterpret_cast<GLADloadproc>(glfwGetProcAddress)) ==
        0) {
        std::cerr << "Failed to initialize GLAD\n";
        return 1;
    }

    // set the viewport dimensions?
    glViewport(0, 0, SCR_WIDTH, SCR_HEIGHT);

    // set callbacks for window resizes and keystrokes
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwSetKeyCallback(window, key_handler);

    // ===
    // === SHADER STUFFS (Rect VAO, VBO, EBO)
    // ===
    float positions[] = {
        0.5F,  0.5F,  0.0F,  // top right
        0.5F,  -0.5F, 0.0F,  // bottom right
        -0.5F, -0.5F, 0.0F,  // bottom left
        -0.5F, 0.5F,  0.0F   // top left
    };

    float colors[] = {
        1.0F, 0.0F, 0.0F,  // top right
        0.0F, 1.0F, 0.0F,  // top right
        0.0F, 0.0F, 1.0F,  // top right
        1.0F, 1.0F, 1.0F   // top right
    };

    uint indices[] = {
        // note that we start from 0!
        0, 1, 3,  // first triangle
        1, 2, 3   // second triangle
    };

    // setup and bind vertex array object
    uint rectangle_VAO;
    glGenVertexArrays(1, &rectangle_VAO);
    glBindVertexArray(rectangle_VAO);

    // setup vertex buffer object
    // with our vertices
    uint rectangle_positions_VBO;
    glGenBuffers(1, &rectangle_positions_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_positions_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(positions),
                 static_cast<GLvoid*>(positions), GL_STATIC_DRAW);

    // define the location and format of the vertex position attribute,
    // index=0, b/c we said location=0
    // 3 b/c 3 values,
    // GL_FALSE b/c we don't need normalization,
    // 3*floatsize is stride, (0 means packed, equivalent in this case)
    // first coord is at [0]
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, nullptr);

    // setup some colors too
    uint rectangle_colors_VBO;
    glGenBuffers(1, &rectangle_colors_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_colors_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(colors),
                 static_cast<GLvoid*>(colors), GL_STATIC_DRAW);

    // similar format for the colors
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 0, nullptr);

    // setup the element buffer object
    uint rectangle_points_EBO = 0;
    glGenBuffers(1, &rectangle_points_EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, rectangle_points_EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices),
                 static_cast<GLvoid*>(indices), GL_STATIC_DRAW);

    // enable the vertex attrib arrays?
    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);

    ShaderProgram leShaderProgram{
        ShaderUnit<GL_FRAGMENT_SHADER>{File{"./shaders/fragment.glsl"}},
        ShaderUnit<GL_VERTEX_SHADER>{File{"./shaders/vertex.glsl"}}
    };

    // ===
    // === TEXTURE BINDING STUFFS (BOX)
    // ===
    unsigned int boxTexture;
    glGenTextures(1, &boxTexture);

    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, boxTexture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    {
        int width, height, nrChannels;
        stbi_set_flip_vertically_on_load(static_cast<int>(true));
        uint8_t* data = stbi_load("./textures/container.jpg", &width, &height, &nrChannels, 0);
        if (data != nullptr) {
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, width, height, 0, GL_RGB, GL_UNSIGNED_BYTE, data);
            glGenerateMipmap(GL_TEXTURE_2D); // probably not needed for later lol
        } else {
            std::cerr << "Failed to load texture\n";
        }

        stbi_image_free(data);
    }

    // ===
    // === TEXTURE BINDING STUFFS (SMILEY)
    // ===
    unsigned int smileyTexture;
    glGenTextures(1, &smileyTexture);

    glActiveTexture(GL_TEXTURE1);
    glBindTexture(GL_TEXTURE_2D, smileyTexture);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    {
        int width, height, nrChannels;
        stbi_set_flip_vertically_on_load(static_cast<int>(true));
        uint8_t* data = stbi_load("./textures/awesomeface.png", &width, &height, &nrChannels, 0);
        if (data != nullptr) {
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, data);
            glGenerateMipmap(GL_TEXTURE_2D); // probably not needed for later lol
        } else {
            std::cerr << "Failed to load texture\n";
        }

        stbi_image_free(data);
    }

    // ===
    // === TEXTURE VBO STUFFS
    // ===
    float tex_coords[] = {
        1.0F,  1.0F, // top right
        1.0F,  0.0F, // bottom right
        0.0F, 0.0F, // bottom left
        0.0F, 1.0F, // top left
    };

    uint rectangle_tex_VBO;
    glGenBuffers(1, &rectangle_tex_VBO);
    glBindBuffer(GL_ARRAY_BUFFER, rectangle_tex_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(tex_coords),
                 static_cast<GLvoid*>(tex_coords), GL_STATIC_DRAW);

    // similar format for the colors
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 0, nullptr);
    glEnableVertexAttribArray(2);

    // ===
    // === CUDA TEXTURE STUFFS
    // ===

    // Create an OpenGL texture
    GLuint texture_id;
    glGenTextures(1, &texture_id);
    glActiveTexture(GL_TEXTURE2);
    glBindTexture(GL_TEXTURE_2D, texture_id);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, CU_TEX_WIDTH, CU_TEX_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    // Register the texture with CUDA
    hipGraphicsResource* cuda_texture_resource;
    checkCudaErrors(hipGraphicsGLRegisterImage(&cuda_texture_resource, texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));

    // install shader and set uniforms so we can tell samplers' offsets ig?
    leShaderProgram.glUseProgram();
    leShaderProgram.glUniform("box", 0);
    leShaderProgram.glUniform("smiley", 1);
    leShaderProgram.glUniform("cuda", 2);
    glUseProgram(0);

    // ===
    // === RENDER LOOP
    // ===
    while (glfwWindowShouldClose(window) == 0) {
        GLCHECK(); // justin casey's

        auto glfwTime = static_cast<float>(glfwGetTime());

        glClearColor(0.2F, 0.3F, 0.3F, 1.0F);
        glClear(GL_COLOR_BUFFER_BIT);

        // ===
        // ===
        // ===

        // Map the cuda texture to CUDA
        hipArray* cuda_texture_array;
        checkCudaErrors(hipGraphicsMapResources(1, &cuda_texture_resource));
        checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&cuda_texture_array, cuda_texture_resource, 0, 0));

         // Create a surface object
        hipResourceDesc res_desc;
        memset(&res_desc, 0, sizeof(res_desc));
        res_desc.resType = hipResourceTypeArray;
        res_desc.res.array.array = cuda_texture_array;
        hipSurfaceObject_t output_surface;
        checkCudaErrors(hipCreateSurfaceObject(&output_surface, &res_desc));

        // === Run the CUDA kernel
        dim3 block(16, 16);
        dim3 grid((CU_TEX_WIDTH + block.x - 1) / block.x, (CU_TEX_HEIGHT + block.y - 1) / block.y);
        CameraInfo* camInfo;
        checkCudaErrors(hipMallocManaged(&camInfo, sizeof(CameraInfo)));
        memset(camInfo, 0, sizeof(CameraInfo));
        // TODO: get this from current reso or summink?
        camInfo->imageResolution = make_uint2(CU_TEX_WIDTH, CU_TEX_HEIGHT);
        camInfo->fovDegrees = 90;
        write_texture_kernel<<<grid, block>>>(output_surface, camInfo, glfwTime);
        checkCudaErrors(hipDeviceSynchronize());

        // Unmap the texture so that OpenGL can use it
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_texture_resource));

        // ===
        // ===
        // ===

        // install the shader program and draw stuffs
        leShaderProgram.glUseProgram();

        // pass time to the shaders so we can have a fiesta
        leShaderProgram.glUniform("glfwTime", glfwTime);

        glBindVertexArray(rectangle_VAO);
        // glDrawArrays(GL_TRIANGLES, 0, 3); // draw 3 verts
        glDrawElements(GL_TRIANGLES, sizeof(indices), GL_UNSIGNED_INT,
                       nullptr);  // draw using ebo
        glBindVertexArray(0);     // unbind, no need to unbind it every time tho

        glfwSwapBuffers(window);
        glfwPollEvents();

#ifdef PAUSE_FRAMES
        getchar(); // tmp boonk for going frame by frame
#endif // PAUSE_FRAMES
    }

    // cleanup a little and exit
    glDeleteVertexArrays(1, &rectangle_VAO);
    glDeleteBuffers(1, &rectangle_positions_VBO);
    glDeleteBuffers(1, &rectangle_points_EBO);
    glfwDestroyWindow(window);
    glfwTerminate();
}
